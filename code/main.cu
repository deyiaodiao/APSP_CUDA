#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdlib> // 包含随机数生成函数
#include <ctime>   // 包含时间函数，用于设置随机数种子
using namespace std;

void test();
void single_test(char* graphfile, int m_size, int*, bool);
void printMatrix(float* matrix, int rows, int cols);

float* initializeMatrix( int N) {
    // 设置随机数种子
    srand(time(NULL));
    
    float* in_dist;
    in_dist = (float*) malloc(sizeof(float)*N*N);
    // 遍历矩阵每个元素
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (i == j) {
                // 对角线元素为0
                in_dist[i * N + j] = 0.0f;
            } else {
                // 其余元素为随机值，这里简单地生成[0, 1)之间的随机值
                in_dist[i * N + j] = static_cast<float>(rand()) / RAND_MAX;
            }
        }
    }
    return in_dist;
}


int main(int argc, char* argv[])
{   
    test();
}

void test()
{
    // int n_vertex[10] = {2,10,100,200,500,1000,2000,5000,10000,20000};
    int n_vertex[3] = {2,3,4};
    char buffer[200];
    int times[2] = {0,0};
    ofstream results;
    results.open("unroll.txt");
    for (int n=0; n<sizeof(n_vertex)/sizeof(n_vertex[0]); n++){
        sprintf(buffer,"../data/graph_5_%d.txt", n_vertex[n]);
        single_test(buffer, n_vertex[n], times, false);
        results<< n_vertex[n] <<" " <<times[0]<<" "<<times[1]<<endl;
    }
}

void single_test(char* graphfile, int m_size, int* times, bool gpu_only)
{
    cout << "load: " << graphfile << endl;
    // load graph adjacent matrix and modify zero to float max
    //cpu matrix
    float* in_dist;
    // int* in_path;
    float* out_dist;
    // int* out_path;
    auto start_load = chrono::steady_clock::now();
    // in_dist = loadGraph(graphfile, m_size);
    in_dist=initializeMatrix(m_size);
    printMatrix(in_dist,m_size,m_size);
    auto end_load = chrono::steady_clock::now();
    auto elapsed_load = chrono::duration_cast<chrono::milliseconds>(end_load - start_load);
    cout<<"graph load time: "<<elapsed_load.count()<<endl;

    
    
    hipEvent_t start_GPU, stop_GPU;
    float gpu_time;
    hipEventCreate(&start_GPU);
    hipEventCreate(&stop_GPU);
    hipEventRecord(start_GPU, 0);

    float* in_dist_d;
    float* out_dist_d;
    out_dist_d = (float*) malloc(sizeof(float)*m_size*m_size);





    hipMalloc((void**)&in_dist_d, sizeof(float)*m_size*m_size);
    hipMemcpy(in_dist_d, in_dist, sizeof(float)*m_size*m_size,
                hipMemcpyHostToDevice);

    // float* print_dist_d;
    // print_dist_d = (float*) malloc(sizeof(float)*m_size*m_size);
    // hipMemcpy(print_dist_d, in_dist_d, sizeof(float)*m_size*m_size, 
    //     hipMemcpyDeviceToHost);
    // cout << "GPU input -------------------------------------------" << endl;
    //     printMatrix(print_dist_d,m_size,m_size);


    // cuda compute
    
    cuda_apsp(in_dist_d, m_size);
    
    hipEventRecord(stop_GPU, 0);
    hipEventSynchronize(stop_GPU);
    hipEventElapsedTime(&gpu_time, start_GPU, stop_GPU);
    hipEventDestroy(start_GPU);
    hipEventDestroy(stop_GPU);
    
    // copy result to host
    hipMemcpy(out_dist_d, in_dist_d, sizeof(float)*m_size*m_size, 
                hipMemcpyDeviceToHost);

    out_dist = (float*) malloc(sizeof(float)*m_size*m_size);
    
    if (!gpu_only){
        // run apsp in cpu and record time
        auto start = chrono::steady_clock::now();
        computeGold(out_dist, in_dist,  m_size);

        auto end = chrono::steady_clock::now();
        auto elapsed = chrono::duration_cast<chrono::milliseconds>(end - start);
        
        // verity the correctness of gpu result
        bool res = correct(out_dist, out_dist_d, m_size, 0.00001);
        
        if (res) cout<<"test pass!"<<endl;
        else cout<<"test fail!!"<<endl;
        //elapsed.count();
        cout << "CPU time " << elapsed.count() << " milliseconds." << endl;
        times[0] = elapsed.count();
    }
    else
        times[0] = 0.0;
    cout << "GPU time " << gpu_time << " milliseconds." << endl;
    cout << "GPU input -------------------------------------------" << endl;
    printMatrix(in_dist,m_size,m_size);
    cout << "GPU result -------------------------------------------" << endl;
    printMatrix(out_dist_d,m_size,m_size);
    // free memory

    free(in_dist);          in_dist=NULL;
    free(out_dist);         out_dist=NULL;
    free(out_dist_d);       out_dist_d=NULL;
    hipFree(in_dist_d);    in_dist_d=NULL;

    times[1] = gpu_time;
    return;
}